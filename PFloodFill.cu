
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdio.h>


#define SIZE 10


// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;
void starttime() {
  gettimeofday( &start, 0 );
}

void endtime(const char* c) {
   gettimeofday( &end, 0 );
   double elapsed = ( end.tv_sec - start.tv_sec ) * 1000.0 + ( end.tv_usec - start.tv_usec ) / 1000.0;
   printf("%s: %f ms\n", c, elapsed);

}


void flood_fill (int nodeX, int nodeY, int map[SIZE][SIZE])
{
   
    int target = map[nodeX][nodeY];
    if (nodeX < 0) return;
    if (nodeX >= SIZE) return;
    if (nodeY < 0) return;
    if (nodeY >= SIZE) return;
    if (target == 1 ) return;
    if (target != 0) return;
    map[nodeX][nodeY] = 2;


    flood_fill (nodeX + 1, nodeY, map); // East
    flood_fill (nodeX, nodeY + 1, map); // South
    flood_fill (nodeX - 1, nodeY, map); // West
    flood_fill (nodeX, nodeY - 1, map); // North

    return;       
}


void print_array2D (int toPrint[SIZE][SIZE])
{
    for(int i = 0; i < 10; i++)
    {
        for(int j = 0; j < 10; j++)
        {
            printf("%d", toPrint[i][j]);
        }
        printf("\n");
    }

}


int main()
{
    int map [SIZE][SIZE] = {
        {0,0,0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0,0,0},
        {1,1,1,1,1,1,1,1,1,0},
        {0,0,0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0,0,0},
        {0,0,0,0,0,0,0,0,0,0}
    };

    starttime();
    flood_fill(0, 0, map);
    endtime("Normal");
    print_array2D(map);
    return 0;
}

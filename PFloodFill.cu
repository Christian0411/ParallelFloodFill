
#include <hip/hip_runtime.h>
#include <stdio.h>

void flood_fill (int nodeX, int nodeY, int map[6][6])
{
   
    int target = map[nodeX][nodeY];
    if (nodeX < 0) return;
    if (nodeX >= 6) return;
    if (nodeY < 0) return;
    if (nodeY >= 6) return;
    if (target == 1 ) return;
    if (target != 0) return;
    map[nodeX][nodeY] = 2;


    flood_fill (nodeX + 1, nodeY, map); // East
    flood_fill (nodeX, nodeY + 1, map); // South
    flood_fill (nodeX - 1, nodeY, map); // West
    flood_fill (nodeX, nodeY - 1, map); // North

    return;       
}


void print_array2D (int toPrint[6][6])
{
    for(int i = 0; i < 6; i++)
    {
        for(int j = 0; j < 6; j++)
        {
            printf("%d", toPrint[i][j]);
        }
        printf("\n");
    }

}


int main()
{
    int map [6][6] = {
        {0,0,0,0,0,0},
        {0,1,1,1,1,0},
        {0,1,0,0,1,0},
        {0,1,0,0,1,0},
        {0,1,1,1,1,0},
        {0,0,0,0,0,0}
    };

    flood_fill(0, 0, map);
    print_array2D(map);
    return 0;
}
